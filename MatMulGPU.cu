#include "hip/hip_runtime.h"
//#include <shrUtils.h>
//#include <shrQATest.h>
#include <sdkHelper.h>
#include "cutil_inline.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<cstdlib>
#include<dos.h>
#include<conio.h>
#include<iostream>
#define MIN 0
#define MAX 1024
#define BLOCK_SIZE 16
using namespace std;
int val=512;
FILE *fp=fopen("data.txt","w");
typedef struct 
{ 
	int width; 
	int height; 
	float* elements; 
} Matrix;
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C) 
{ 

	// Each thread computes one element of C // by accumulating results into Cvalue 
	float Cvalue = 0; 
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x; 
	for (int e = 0; e < A.width; ++e) 
	Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col]; 
	C.elements[row * C.width + col] = Cvalue; 
}
void MatMul(const Matrix A, const Matrix B, Matrix C) 
{ 
	// Load A and B to device memory 
	Matrix d_A; 
	d_A.width = A.width; 
	d_A.height = A.height; 
	size_t size = A.width * A.height * sizeof(float); 
	hipMalloc(&d_A.elements, size); 
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice); 
	Matrix d_B; 
	d_B.width = B.width; 
	d_B.height = B.height; 
	size = B.width * B.height * sizeof(float); 
	hipMalloc(&d_B.elements, size); 
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	// Allocate C in device memory 
	Matrix d_C; 
	d_C.width = C.width; 
	d_C.height = C.height; 
	size = C.width * C.height * sizeof(float); 
	hipMalloc(&d_C.elements, size); 
	// Invoke kernel 
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y); 
	StopWatchInterface * timer_mul=NULL;
	sdkCreateTimer(&timer_mul);
	sdkStartTimer(&timer_mul);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C); 
	// Read C from device memory 
	sdkStopTimer(&timer_mul);
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost); 
	
	float dSeconds = sdkGetTimerValue(&timer_mul);
	//cout<<"Matrix size : "<<val<<"x"<<val<<endl;
	//cout<<"Multiplication Time = "<<dSeconds<<"ms"<<endl;
	printf("%d\t%f\n",val,dSeconds);
	fprintf(fp,"%d\t%f\n",val,dSeconds);
	//getch();
	sdkDeleteTimer(&timer_mul);
	// Free device memory 
	hipFree(d_A.elements); 
	hipFree(d_B.elements); 
	hipFree(d_C.elements); 
}
int main()
{
	Matrix l_a,l_b,l_c;
	int i,j,k=1;
	while(val<=12300)
	{
		l_a.width=l_a.height=l_b.width=l_b.height=l_c.width=l_c.height=val;
		l_a.elements=(float*)malloc(l_a.width*l_a.height*sizeof(float));
		l_b.elements=(float*)malloc(l_b.width*l_b.height*sizeof(float));
		l_c.elements=(float*)malloc(l_c.width*l_c.height*sizeof(float));
		for(i=MIN;i<val;i++)
		{
                     for(j=MIN;j<val;j++)
                     {
                                      l_a.elements[i*val+j]=(rand() / (float)RAND_MAX);
                                      l_b.elements[i*val+j]=(rand() / (float)RAND_MAX);
                     }
		}
		MatMul(l_a,l_b,l_c);
		free(l_a.elements);
		free(l_b.elements);
		free(l_c.elements);
		val=val+512;
		//else
			//val=val*2;
		//k++;
	}
	fclose(fp);
	getch();
    return 0;
}
